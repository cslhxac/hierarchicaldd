#include "hip/hip_runtime.h"
//#####################################################################
//Removed For Anonymity: Copyright Authors of Submission pap173s1
// Distributed under the FreeBSD license (see license.txt)
//#####################################################################
#include <iostream>
#include "Smoother_In_Cache_Helper_CUDA.h"
#include <SPGrid_Fluids/Solvers/Domain_Decomposition/SPGRID_MULTIGRID_FLAGS.h>
using namespace SPGrid;
#define THREADBLOCK 1024
#define BLOCKS_IN_CACHE 10
namespace{
template<unsigned d> struct BitCount;
template<> struct BitCount<0>  {enum {value=0};};
template<unsigned d> struct BitCount {enum {value=(d&1)+BitCount<(d>>1)>::value};};
__device__ float Dinv[64] = {  0.f, 1.f/(float)BitCount< 1>::value, 1.f/(float)BitCount< 2>::value, 1.f/(float)BitCount< 3>::value, 
    1.f/(float)BitCount< 4>::value, 1.f/(float)BitCount< 5>::value, 1.f/(float)BitCount< 6>::value, 1.f/(float)BitCount< 7>::value, 
    1.f/(float)BitCount< 8>::value, 1.f/(float)BitCount< 9>::value, 1.f/(float)BitCount<10>::value, 1.f/(float)BitCount<11>::value, 
    1.f/(float)BitCount<12>::value, 1.f/(float)BitCount<13>::value, 1.f/(float)BitCount<14>::value, 1.f/(float)BitCount<15>::value, 
    1.f/(float)BitCount<16>::value, 1.f/(float)BitCount<17>::value, 1.f/(float)BitCount<18>::value, 1.f/(float)BitCount<19>::value,        
    1.f/(float)BitCount<20>::value, 1.f/(float)BitCount<21>::value, 1.f/(float)BitCount<22>::value, 1.f/(float)BitCount<23>::value, 
    1.f/(float)BitCount<24>::value, 1.f/(float)BitCount<25>::value, 1.f/(float)BitCount<26>::value, 1.f/(float)BitCount<27>::value, 
    1.f/(float)BitCount<28>::value, 1.f/(float)BitCount<29>::value, 1.f/(float)BitCount<30>::value, 1.f/(float)BitCount<31>::value, 
    1.f/(float)BitCount<32>::value, 1.f/(float)BitCount<33>::value, 1.f/(float)BitCount<34>::value, 1.f/(float)BitCount<35>::value, 
    1.f/(float)BitCount<36>::value, 1.f/(float)BitCount<37>::value, 1.f/(float)BitCount<38>::value, 1.f/(float)BitCount<39>::value,
    1.f/(float)BitCount<40>::value, 1.f/(float)BitCount<41>::value, 1.f/(float)BitCount<42>::value, 1.f/(float)BitCount<43>::value, 
    1.f/(float)BitCount<44>::value, 1.f/(float)BitCount<45>::value, 1.f/(float)BitCount<46>::value, 1.f/(float)BitCount<47>::value, 
    1.f/(float)BitCount<48>::value, 1.f/(float)BitCount<49>::value, 1.f/(float)BitCount<50>::value, 1.f/(float)BitCount<51>::value, 
    1.f/(float)BitCount<52>::value, 1.f/(float)BitCount<53>::value, 1.f/(float)BitCount<54>::value, 1.f/(float)BitCount<55>::value, 
    1.f/(float)BitCount<56>::value, 1.f/(float)BitCount<57>::value, 1.f/(float)BitCount<58>::value, 1.f/(float)BitCount<59>::value,
    1.f/(float)BitCount<60>::value, 1.f/(float)BitCount<61>::value, 1.f/(float)BitCount<62>::value, 1.f/(float)BitCount<63>::value
};
}
template <class T, int log2_struct,int xsize,int ysize,int zsize,class T_offset_ptr>
__global__ void Bottom_Smoother_In_Cache_Kernel_3D(char* const data,
                                                   T_offset_ptr mask_offset,T_offset_ptr r_offset,
                                                   T_offset_ptr u_offset,T_offset_ptr rhs_offset,
                                                   const T_offset_ptr* offsets,
                                                   const T_offset_ptr* const b_x_minus,
                                                   const T_offset_ptr* const b_x_plus,
                                                   const T_offset_ptr* const b_y_minus,
                                                   const T_offset_ptr* const b_y_plus,
                                                   const T_offset_ptr* const b_z_minus,
                                                   const T_offset_ptr* const b_z_plus,
                                                   const int max_block,const T omega,
                                                   const int iterations){
    enum {
        DATABLOCK=xsize*ysize*zsize,
        span=THREADBLOCK/DATABLOCK,
        xstride=ysize*zsize,
        ystride=zsize,
        zstride=1,
        page_size=4096
    };
    const unsigned int block = threadIdx.x / DATABLOCK;
    const unsigned int entry = threadIdx.x % DATABLOCK;
    const unsigned int z = entry % zsize;
    const unsigned int y = entry / zsize % ysize;
    const unsigned int x = entry / zsize / ysize;
    typedef SPGrid_Mask<log2_struct, NextLogTwo<sizeof(T)>::value,3> T_MASK;

    __shared__ T_offset_ptr block_index[BLOCKS_IN_CACHE];
    __shared__ T_offset_ptr block_minus_x_index[BLOCKS_IN_CACHE];
    __shared__ T_offset_ptr block_plus_x_index[BLOCKS_IN_CACHE];
    __shared__ T_offset_ptr block_minus_y_index[BLOCKS_IN_CACHE];
    __shared__ T_offset_ptr block_plus_y_index[BLOCKS_IN_CACHE];
    __shared__ T_offset_ptr block_minus_z_index[BLOCKS_IN_CACHE];
    __shared__ T_offset_ptr block_plus_z_index[BLOCKS_IN_CACHE];
    __shared__ float Dinv_buffered[64];
    __shared__ char data_buffer[(BLOCKS_IN_CACHE+1)*page_size];
    if(threadIdx.x < 64)
        Dinv_buffered[threadIdx.x] = Dinv[threadIdx.x];
    if(threadIdx.x < BLOCKS_IN_CACHE)
        if(threadIdx.x < max_block){
            block_index[threadIdx.x] = offsets[threadIdx.x];
            block_minus_x_index[threadIdx.x]=b_x_minus[threadIdx.x];
            block_plus_x_index[threadIdx.x]=b_x_plus[threadIdx.x];
            block_minus_y_index[threadIdx.x]=b_y_minus[threadIdx.x];
            block_plus_y_index[threadIdx.x]=b_y_plus[threadIdx.x];
            block_minus_z_index[threadIdx.x]=b_z_minus[threadIdx.x];
            block_plus_z_index[threadIdx.x]=b_z_plus[threadIdx.x];
        }
    for(int i=0;i<(BLOCKS_IN_CACHE+1)*DATABLOCK;i+=THREADBLOCK){
        (reinterpret_cast<T*>(data_buffer))[i*THREADBLOCK+threadIdx.x]=(reinterpret_cast<T*>(data))[i*THREADBLOCK+threadIdx.x];}
    T* r=reinterpret_cast<T*>((unsigned long)data_buffer+r_offset*DATABLOCK);
    T* u=reinterpret_cast<T*>((unsigned long)data_buffer+u_offset*DATABLOCK);
    T* rhs=reinterpret_cast<T*>((unsigned long)data_buffer+rhs_offset*DATABLOCK);
    unsigned* masks=reinterpret_cast<unsigned*>((unsigned long)data_buffer+mask_offset*DATABLOCK);
    for(int itr=0;itr<iterations;++itr){
        __syncthreads();
        for(int i = 0;i < BLOCKS_IN_CACHE;i += span){
            if (i + block < max_block){
                T* r_base = reinterpret_cast<T*>((unsigned long)r + (unsigned long)block_index[i + block]); 
                unsigned mask_value = reinterpret_cast<unsigned*>((unsigned long)masks + (unsigned long)block_index[i + block])[entry];
                if(mask_value & SPGrid_Solver_Cell_Type_Active){
                    T center_value = reinterpret_cast<T*>((unsigned long)u + (unsigned long)block_index[i + block])[entry];
                    T b_value = reinterpret_cast<T*>((unsigned long)rhs + (unsigned long)block_index[i + block])[entry];

                    T& x_minus_value = (x==0)
                        ? reinterpret_cast<T*>((unsigned long)u + (unsigned long)block_minus_x_index[i + block])[entry+(xsize-1)*xstride]
                        : reinterpret_cast<T*>((unsigned long)u + (unsigned long)block_index[i + block])[entry-xstride];
                    T& x_plus_value = (x==xsize-1)
                        ? reinterpret_cast<T*>((unsigned long)u + (unsigned long)block_plus_x_index[i + block])[entry-(xsize-1)*xstride]
                        : reinterpret_cast<T*>((unsigned long)u + (unsigned long)block_index[i + block])[entry+xstride];
                                
                    T& y_minus_value = (y==0)
                        ? reinterpret_cast<T*>((unsigned long)u + (unsigned long)block_minus_y_index[i + block])[entry+(ysize-1)*ystride]
                        : reinterpret_cast<T*>((unsigned long)u + (unsigned long)block_index[i + block])[entry-ystride];
                    T& y_plus_value = (y==ysize-1)
                        ? reinterpret_cast<T*>((unsigned long)u + (unsigned long)block_plus_y_index[i + block])[entry-(ysize-1)*ystride]
                        : reinterpret_cast<T*>((unsigned long)u + (unsigned long)block_index[i + block])[entry+ystride];
                    
                    T& z_minus_value = (z==0)
                        ? reinterpret_cast<T*>((unsigned long)u + (unsigned long)block_minus_z_index[i + block])[entry+(zsize-1)*zstride]
                        : reinterpret_cast<T*>((unsigned long)u + (unsigned long)block_index[i + block])[entry-zstride];
                    T& z_plus_value = (z==zsize-1)
                        ? reinterpret_cast<T*>((unsigned long)u + (unsigned long)block_plus_z_index[i + block])[entry-(zsize-1)*zstride]
                        : reinterpret_cast<T*>((unsigned long)u + (unsigned long)block_index[i + block])[entry+zstride];
                    
                    T result=0;
                    T x_minus = (x_minus_value - center_value);
                    T x_plus = (x_plus_value - center_value);
                    T y_minus = (y_minus_value - center_value);
                    T y_plus = (y_plus_value - center_value);
                    T z_minus = (z_minus_value - center_value);
                    T z_plus = (z_plus_value - center_value);

                    if (mask_value & (SPGrid_Solver_Face_Minus_X_Active))
                        result -= x_minus;
                    if (mask_value & (SPGrid_Solver_Face_Plus_X_Active))
                        result -= x_plus; 

                    if (mask_value & (SPGrid_Solver_Face_Minus_Y_Active))
                        result -= y_minus;
                    if (mask_value & (SPGrid_Solver_Face_Plus_Y_Active))
                        result -= y_plus;
 
                    if (mask_value & (SPGrid_Solver_Face_Minus_Z_Active))
                        result -= z_minus; 
                    if (mask_value & (SPGrid_Solver_Face_Plus_Z_Active))
                        result -= z_plus; 
                                   
                    r_base[entry] = b_value - result;
                }
            }
        }
        __syncthreads();
        for(int i = 0;i < BLOCKS_IN_CACHE;i += span){
            if (i + block < max_block){
                unsigned int cell_flag = reinterpret_cast<unsigned int*>((unsigned long)masks + block_index[i+block])[entry];
                if(cell_flag & SPGrid_Solver_Cell_Type_Active){
                    cell_flag = (cell_flag >> (BitLength<SPGrid_Solver_Face_Minus_X_Active>::value-1))&0x3f;
                    T* u_ptr = &(reinterpret_cast<T*>((unsigned long)u + block_index[i+block])[entry]);
                    const T* r_ptr = &(reinterpret_cast<const T*>((unsigned long)r + block_index[i+block])[entry]);
                    *(u_ptr) = *(u_ptr) + *(r_ptr)*omega*Dinv_buffered[cell_flag];
                }
            }
        }
    }
    //TODO: only copy u channel back.
    for(int i=0;i<(BLOCKS_IN_CACHE+1)*DATABLOCK;i+=THREADBLOCK){
        (reinterpret_cast<T*>(data))[i*THREADBLOCK+threadIdx.x]=(reinterpret_cast<T*>(data_buffer))[i*THREADBLOCK+threadIdx.x];}
}
//#####################################################################
// Function Bottom_Smoothing
//#####################################################################
// T_MASK corresponds to the mask for the data (not the mask channel)
template <class T,int log2_struct,class T_offset_ptr> 
void Smoother_In_Cache_Helper_CUDA<T,log2_struct,3,T_offset_ptr>::Bottom_Smoothing(const unsigned int index_start,
                                                                                   const unsigned int index_end,
                                                                                   hipStream_t& hip_stream)
{
    int number_of_blocks=index_end-index_start+1;
    int number_of_cuda_blocks = (number_of_blocks%BLOCKS_IN_CACHE)?(number_of_blocks/BLOCKS_IN_CACHE+1):(number_of_blocks/BLOCKS_IN_CACHE);
    if(number_of_cuda_blocks == 0) return;
    if(number_of_cuda_blocks>1) {std::cout<<"The smoother in cache kernel only supports single cuda block due to the fact that it requires global synchronization.";exit(1);}
    if(THREADBLOCK>=block_xsize*block_ysize*block_zsize){std::cout<<"The number of cuda thread is more than number of elements per block.";exit(1);}
    Bottom_Smoother_In_Cache_Kernel_3D<T,log2_struct,block_xsize,block_ysize,block_zsize,T_offset_ptr>
        <<<number_of_cuda_blocks,THREADBLOCK,0,hip_stream>>>
        (data_ptr,mask_offset,r_offset,u_offset,rhs_offset,
         b+index_start,
         b_x_minus+index_start,b_x_plus+index_start,
         b_y_minus+index_start,b_y_plus+index_start,
         b_z_minus+index_start,b_z_plus+index_start,
         number_of_blocks,omega,iterations);

    // hipDeviceSynchronize();
    // hipError_t err = hipGetLastError();
    // if(err!=hipSuccess) {std::cout<<"Something went wrong in residual kernel! Msg: "<< hipGetErrorString(err)<<std::endl;abort();}
}
//#####################################################################
template class Smoother_In_Cache_Helper_CUDA<float,4,3,unsigned int>;
