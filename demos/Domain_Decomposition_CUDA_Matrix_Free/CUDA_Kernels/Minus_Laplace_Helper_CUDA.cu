#include "hip/hip_runtime.h"
//#####################################################################
//Removed For Anonymity: Copyright Authors of Submission pap173s1
// Distributed under the FreeBSD license (see license.txt)
//#####################################################################
#include <iostream>
#include "Minus_Laplace_Helper_CUDA.h"
using namespace SPGrid;
#define THREADBLOCK 256
#define PREFETCH 256

template <class T, int log2_struct,int xsize,int ysize,int zsize,class T_offset_ptr,bool accumulative>
__global__ void Minus_Laplace_Kernel_3D(const unsigned* masks, T* out,const T* in,
                                        const T_offset_ptr* offsets,
                                        const T_offset_ptr* const b_x_minus,
                                        const T_offset_ptr* const b_x_plus,
                                        const T_offset_ptr* const b_y_minus,
                                        const T_offset_ptr* const b_y_plus,
                                        const T_offset_ptr* const b_z_minus,
                                        const T_offset_ptr* const b_z_plus,
                                        int max_block,const unsigned flag_to_check){
    enum {
        DATABLOCK=xsize*ysize*zsize,
        span=THREADBLOCK/DATABLOCK,
        xstride=ysize*zsize,
        ystride=zsize,
        zstride=1  
    };
    const unsigned int block = threadIdx.x / DATABLOCK;
    const unsigned int entry = threadIdx.x % DATABLOCK;
    const unsigned int z = entry % zsize;
    const unsigned int y = entry / zsize % ysize;
    const unsigned int x = entry / zsize / ysize;
    typedef SPGrid_Mask<log2_struct, NextLogTwo<sizeof(T)>::value,3> T_MASK;

    __shared__ T_offset_ptr block_index[PREFETCH];
    __shared__ T_offset_ptr block_minus_x_index[PREFETCH];
    __shared__ T_offset_ptr block_plus_x_index[PREFETCH];
    __shared__ T_offset_ptr block_minus_y_index[PREFETCH];
    __shared__ T_offset_ptr block_plus_y_index[PREFETCH];
    __shared__ T_offset_ptr block_minus_z_index[PREFETCH];
    __shared__ T_offset_ptr block_plus_z_index[PREFETCH];
    if(threadIdx.x < PREFETCH)
        if(blockIdx.x * PREFETCH + threadIdx.x < max_block){
            block_index[threadIdx.x] = offsets[blockIdx.x * PREFETCH + threadIdx.x];
            block_minus_x_index[threadIdx.x]=b_x_minus[blockIdx.x * PREFETCH + threadIdx.x];
            block_plus_x_index[threadIdx.x]=b_x_plus[blockIdx.x * PREFETCH + threadIdx.x];
            block_minus_y_index[threadIdx.x]=b_y_minus[blockIdx.x * PREFETCH + threadIdx.x];
            block_plus_y_index[threadIdx.x]=b_y_plus[blockIdx.x * PREFETCH + threadIdx.x];
            block_minus_z_index[threadIdx.x]=b_z_minus[blockIdx.x * PREFETCH + threadIdx.x];
            block_plus_z_index[threadIdx.x]=b_z_plus[blockIdx.x * PREFETCH + threadIdx.x];
        }
    __syncthreads();

    for(int i = 0;i < PREFETCH;i += span){
        if (blockIdx.x * PREFETCH + i + block < max_block){
            T* out_base = reinterpret_cast<T*>((unsigned long)out + (unsigned long)block_index[i + block]); 
            unsigned mask_value = reinterpret_cast<unsigned*>((unsigned long)masks + (unsigned long)block_index[i + block])[entry];
            if(mask_value & flag_to_check){             
                T center_value = reinterpret_cast<T*>((unsigned long)in + (unsigned long)block_index[i + block])[entry];
                
                T& x_minus_value = (x==0)
                    ? reinterpret_cast<T*>((unsigned long)in + (unsigned long)block_minus_x_index[i + block])[entry+(xsize-1)*xstride]
                    : reinterpret_cast<T*>((unsigned long)in + (unsigned long)block_index[i + block])[entry-xstride];
                T& x_plus_value = (x==xsize-1)
                    ? reinterpret_cast<T*>((unsigned long)in + (unsigned long)block_plus_x_index[i + block])[entry-(xsize-1)*xstride]
                    : reinterpret_cast<T*>((unsigned long)in + (unsigned long)block_index[i + block])[entry+xstride];
                                
                T& y_minus_value = (y==0)
                    ? reinterpret_cast<T*>((unsigned long)in + (unsigned long)block_minus_y_index[i + block])[entry+(ysize-1)*ystride]
                    : reinterpret_cast<T*>((unsigned long)in + (unsigned long)block_index[i + block])[entry-ystride];
                T& y_plus_value = (y==ysize-1)
                    ? reinterpret_cast<T*>((unsigned long)in + (unsigned long)block_plus_y_index[i + block])[entry-(ysize-1)*ystride]
                    : reinterpret_cast<T*>((unsigned long)in + (unsigned long)block_index[i + block])[entry+ystride];
                    
                T& z_minus_value = (z==0)
                    ? reinterpret_cast<T*>((unsigned long)in + (unsigned long)block_minus_z_index[i + block])[entry+(zsize-1)*zstride]
                    : reinterpret_cast<T*>((unsigned long)in + (unsigned long)block_index[i + block])[entry-zstride];
                T& z_plus_value = (z==zsize-1)
                    ? reinterpret_cast<T*>((unsigned long)in + (unsigned long)block_plus_z_index[i + block])[entry-(zsize-1)*zstride]
                    : reinterpret_cast<T*>((unsigned long)in + (unsigned long)block_index[i + block])[entry+zstride];
                    
                T result=0;
                if(accumulative) result = out_base[entry];
                
                T x_minus = (x_minus_value - center_value);
                T x_plus = (x_plus_value - center_value);
                T y_minus = (y_minus_value - center_value);
                T y_plus = (y_plus_value - center_value);
                T z_minus = (z_minus_value - center_value);
                T z_plus = (z_plus_value - center_value);

                if (mask_value & (SPGrid_Solver_Face_Minus_X_Active))
                    result += x_minus;
                if (mask_value & (SPGrid_Solver_Face_Plus_X_Active))
                    result += x_plus; 

                if (mask_value & (SPGrid_Solver_Face_Minus_Y_Active))
                    result += y_minus;
                if (mask_value & (SPGrid_Solver_Face_Plus_Y_Active))
                    result += y_plus;
 
                if (mask_value & (SPGrid_Solver_Face_Minus_Z_Active))
                    result += z_minus; 
                if (mask_value & (SPGrid_Solver_Face_Plus_Z_Active))
                    result += z_plus; 
                
                //if(result!=0) printf("here,%f\n",result);
                out_base[entry] = result;
            }
        }
    }
}
//#####################################################################
// Constructor 3D
//#####################################################################
template <class T, int log2_struct,class T_offset_ptr,bool accumulative>
Minus_Laplace_Helper_CUDA<T,log2_struct,3,T_offset_ptr,accumulative>::Minus_Laplace_Helper_CUDA(T* const x_input,const T* const y_input,const unsigned* const mask_input,
                                                                                                const T_offset_ptr* const b_input,
                                                                                                const T_offset_ptr* const b_x_minus_input,
                                                                                                const T_offset_ptr* const b_x_plus_input,
                                                                                                const T_offset_ptr* const b_y_minus_input,
                                                                                                const T_offset_ptr* const b_y_plus_input,
                                                                                                const T_offset_ptr* const b_z_minus_input,
                                                                                                const T_offset_ptr* const b_z_plus_input,
                                                                                                const int size_input,const unsigned flag_to_check_input)
:x(x_input),y(y_input),mask(mask_input),
    b(b_input),
    b_x_minus(b_x_minus_input),b_x_plus(b_x_plus_input),
    b_y_minus(b_y_minus_input),b_y_plus(b_y_plus_input),
    b_z_minus(b_z_minus_input),b_z_plus(b_z_plus_input),
    size(size_input),flag_to_check(flag_to_check_input)
{
}
//#####################################################################
// Function Run_Index_Range
//#####################################################################
// T_MASK corresponds to the mask for the data (not the mask channel)
template <class T,int log2_struct,class T_offset_ptr,bool accumulative> 
void Minus_Laplace_Helper_CUDA<T,log2_struct,3,T_offset_ptr,accumulative>::Run_Index_Range(const unsigned int index_start,
                                                                                           const unsigned int index_end,
                                                                                           hipStream_t& hip_stream)
{
    int number_of_blocks=index_end-index_start+1;
    int number_of_cuda_blocks = (number_of_blocks%PREFETCH)?(number_of_blocks/PREFETCH+1):(number_of_blocks/PREFETCH);
    if(number_of_cuda_blocks == 0) return;
    Minus_Laplace_Kernel_3D<T,log2_struct,block_xsize,block_ysize,block_zsize,T_offset_ptr,accumulative>
        <<<number_of_cuda_blocks,THREADBLOCK,0,hip_stream>>>
        (mask,x,y,
         b+index_start,
         b_x_minus+index_start,b_x_plus+index_start,
         b_y_minus+index_start,b_y_plus+index_start,
         b_z_minus+index_start,b_z_plus+index_start,
         number_of_blocks,flag_to_check);

    // hipDeviceSynchronize();
    // hipError_t err = hipGetLastError();
    // if(hipSuccess != err){
    //     std::cerr << "Error in Minus Laplace Helper. Msg: "<< hipGetErrorString(err) << std::endl;
    //     abort();
    // }
}
//#####################################################################
template class Minus_Laplace_Helper_CUDA<float,4,3,unsigned int,true>;
